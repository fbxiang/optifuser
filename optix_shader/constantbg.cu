#include "hip/hip_runtime.h"
#include <optix_device.h>

#include "prd.h"

rtDeclareVariable(float3,     bg_color,    ,          );
rtDeclareVariable(PerRayData, current_prd, rtPayload, );

RT_PROGRAM void miss() {
  current_prd.result = bg_color;
  current_prd.done = 1;
}
