#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_math.h>

#include "prd.h"

rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(PerRayData, current_prd, rtPayload, );
RT_PROGRAM void miss() {
  // float theta = atan2f(current_prd.direction.x, current_prd.direction.z);
  float theta = atan2f(-current_prd.direction.y, -current_prd.direction.x);
  float phi = M_PIf * 0.5f - acosf(current_prd.direction.z);
  float u = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v = 0.5f * (1.0f + sin(phi));
  current_prd.radiance = make_float3(tex2D(envmap, u, 1-v));
  current_prd.done = 1;
}
