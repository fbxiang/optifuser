#include "hip/hip_runtime.h"
#include "prd.h"
#include "random.h"
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

// Scene
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(rtObject, top_object, , );

rtDeclareVariable(unsigned int, pathtrace_ray_type, , );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, , );
rtDeclareVariable(float3, bad_color, , );
rtDeclareVariable(unsigned int, n_samples_sqrt, , );
rtDeclareVariable(unsigned int, iterations, , );
rtDeclareVariable(unsigned int, n_rays, , );

rtBuffer<float4, 2> output_buffer;
rtBuffer<float4, 2> albedo_buffer;
rtBuffer<float4, 2> normal_buffer;

rtDeclareVariable(Matrix3x3, normal_matrix, , );

// Camera
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

// Ray
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData, current_prd, rtPayload, );

RT_PROGRAM void camera() {
  size_t2 screen = output_buffer.size();
  float2 inv_screen = 1.f / make_float2(screen) * 2.f;
  float2 pixel = make_float2(launch_index) * inv_screen - 1.f;

  float3 result = make_float3(0.f);
  float3 albedo = make_float3(0.f);
  float3 normal = make_float3(0.f);

  unsigned int seed = tea<16>(screen.x * launch_index.y + launch_index.x, iterations);

  const unsigned int n_samples = n_samples_sqrt * n_samples_sqrt;
  unsigned int n_samples2 = n_samples;

  do {
    float r1, r2;
    halton2d(r1, r2, n_samples2 + iterations * n_samples, 3, 5);

    // pixel is in range [-1, 1) x [-1, 1)
    float2 d = pixel + make_float2(r1, r2) * inv_screen;
    float3 ray_origin = eye;
    float3 ray_direction = normalize(d.x * U + d.y * V + W);

    PerRayData prd;
    prd.albedo = make_float3(0.f);
    prd.normal = make_float3(0.f);
    prd.radiance = make_float3(0.f);
    prd.direction = ray_direction;
    prd.result = make_float3(0.f);
    prd.attenuation = make_float3(1.f);
    prd.done = false;
    prd.depth = 0;
    prd.seed = seed;
    prd.max_depth_override = 0;

    for (;;) {
      float3 attenuation = prd.attenuation;
      Ray ray =
          make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
      rtTrace(top_object, ray, prd);
      prd.result += prd.radiance * attenuation;

      if (prd.done) {
        break;
      }
      prd.depth++;

      if (prd.depth >= prd.max_depth_override && prd.depth >= n_rays) {
        break;
      }

      ray_origin = prd.origin;
      ray_direction = prd.direction;
    }
    result += prd.result;
    albedo += prd.albedo;

    float3 normal_eyespace = (length(prd.normal) > 0.f) ? normalize(normal_matrix * prd.normal)
                                                        : make_float3(0., 0., 1.);
    normal += normal_eyespace;
    seed = prd.seed;

  } while (--n_samples2);

  result /= n_samples;
  albedo /= n_samples;
  normal = normalize(normal);

  float r = 1.f / (iterations + 1);

  output_buffer[launch_index] =
      output_buffer[launch_index] * (1 - r) + make_float4(result, 1.f) * r;
  albedo_buffer[launch_index] =
      albedo_buffer[launch_index] * (1 - r) + make_float4(albedo, 1.f) * r;
  normal_buffer[launch_index] =
      normal_buffer[launch_index] * (1 - r) + make_float4(normal, 1.f) * r;
}

RT_PROGRAM void exception() { output_buffer[launch_index] = make_float4(bad_color, 1.f); }
