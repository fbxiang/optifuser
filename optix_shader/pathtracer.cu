#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include <optix_device.h>
#include "random.h"
#include "prd.h"

// Scene
rtDeclareVariable(float,        scene_epsilon,             ,              );
rtDeclareVariable(uint2,        launch_index,              rtLaunchIndex, );
rtDeclareVariable(rtObject,     top_object,                ,              );

rtDeclareVariable(unsigned int, pathtrace_ray_type,        ,              );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, ,              );
rtDeclareVariable(float3,       bad_color,                 ,              );
rtDeclareVariable(unsigned int, n_samples_sqrt,            ,              );
rtDeclareVariable(unsigned int, iterations,                ,              );
rtDeclareVariable(unsigned int, n_rays,                    ,              );

rtBuffer<float4, 2> output_buffer;


// Camera
rtDeclareVariable(float3 ,      eye ,                      ,              );
rtDeclareVariable(float3 ,      U   ,                      ,              );
rtDeclareVariable(float3 ,      V   ,                      ,              );
rtDeclareVariable(float3 ,      W   ,                      ,              );


// Ray
rtDeclareVariable(optix::Ray,   ray,                       rtCurrentRay,  );
rtDeclareVariable(PerRayData,   current_prd,               rtPayload,     );

RT_PROGRAM void camera() {
  size_t2 screen = output_buffer.size();
  float2 inv_screen = 1.f / make_float2(screen) * 2.f;
  float2 pixel = make_float2(launch_index) * inv_screen - 1.f;

  float3 result = make_float3(0.0f);

  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, iterations);

  float2 jitter_scale = inv_screen / n_samples_sqrt;
  const unsigned int n_samples = n_samples_sqrt * n_samples_sqrt;
  unsigned int n_samples2 = n_samples;

  do {
    float r1, r2;
    halton2d(r1, r2, n_samples2 + iterations * n_samples, 3, 5);

    // pixel is in range [-1, 1) x [-1, 1)
    float2 d             = pixel + make_float2(r1, r2) * inv_screen;
    float3 ray_origin    = eye;
    float3 ray_direction = normalize(d.x * U + d.y * V + W);

    PerRayData prd;
    prd.radiance    = make_float3(0.f);
    prd.direction   = ray_direction;
    prd.result      = make_float3(0.f);
    prd.attenuation = make_float3(1.f);
    prd.done        = false;
    prd.depth       = 0;
    prd.seed        = seed;
    prd.max_depth_override = 0;

    for(;;) {
      Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
      rtTrace(top_object, ray, prd);
      prd.result += prd.radiance * prd.attenuation;

      if (prd.done) {
        break;
      }
      prd.depth++;

      if (prd.depth >= prd.max_depth_override && prd.depth >= n_rays) {
        break;
      }

      ray_origin = prd.origin;
      ray_direction = prd.direction;
    }
    result += prd.result;
    seed = prd.seed;

  } while (--n_samples2);

  result /= n_samples;

  float r = 1.f / (iterations+1);
  float4 color = make_float4(result, 1.f);

  output_buffer[launch_index] = output_buffer[launch_index] * (1-r) + color * r;
}


RT_PROGRAM void exception() {
  output_buffer[launch_index] = make_float4(bad_color, 1.f);
}
