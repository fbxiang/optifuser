#include "hip/hip_runtime.h"
#include "light.h"
#include "prd.h"
#include "random.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, , );

rtBuffer<DirectionalLight> directional_lights;
rtBuffer<PointLight> point_lights;
rtBuffer<ParallelogramLight> parallelogram_lights;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float4, kd, , );
rtDeclareVariable(int, has_kd_map, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, hit_point, attribute hit_point, );

rtDeclareVariable(int, use_shadow, , );

rtTextureSampler<float4, 2> kd_map;

rtDeclareVariable(PerRayData, current_prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void closest_hit() {
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal =
      normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
  float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);

  current_prd.max_depth_override = 0;

  float3 kd_val = make_float3(kd);
  float alpha = kd.w;
  if (has_kd_map) {
    float4 texture_kd_val = tex2D(kd_map, texcoord.x, texcoord.y);
    kd_val = make_float3(texture_kd_val);
    alpha = texture_kd_val.w;
  }

  float3 result = make_float3(0.f);
  if (rnd(current_prd.seed) < alpha) {
    // reflection
    current_prd.origin = hitpoint;
    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    Onb onb(ffnormal);
    onb.inverse_transform(p);
    current_prd.direction = normalize(p);
    current_prd.attenuation *= kd_val;

    for (int i = 0; i < directional_lights.size(); i++) {
      DirectionalLight light = directional_lights[i];
      const float3 L = -light.direction;
      const float nDl = dot(ffnormal, L);
      if (nDl > 0.f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        shadow_prd.inShadow = false;
        Ray shadow_ray =
            make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_shadower, shadow_ray, shadow_prd);

        if (!shadow_prd.inShadow) {
          result += nDl * light.emission * kd_val * shadow_prd.attenuation;
        }
      }
    }

    for (int i = 0; i < point_lights.size(); i++) {
      PointLight light = point_lights[i];
      const float Ldist = length(light.position - hitpoint);
      const float3 L = normalize(light.position - hitpoint);
      const float nDl = dot(ffnormal, L);
      if (nDl > 0.f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        shadow_prd.inShadow = false;
        Ray shadow_ray =
            make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);

        if (!shadow_prd.inShadow) {
          result += kd_val * nDl * light.emission / Ldist / Ldist * shadow_prd.attenuation;
        }
      }
    }
    for (int i = 0; i < parallelogram_lights.size(); ++i) {
      ParallelogramLight light = parallelogram_lights[i];

      // sample a point on the light
      float r1 = rnd(current_prd.seed);
      float r2 = rnd(current_prd.seed);
      float3 light_position = light.corner + r1 * light.v1 + r2 * light.v2;
      const float Ldist = length(light_position - hitpoint);
      const float3 L = normalize(light_position - hitpoint);
      const float nDl = dot(ffnormal, L);
      const float lnDl = dot(light.normal, L); // light normal dot light direction

      if (nDl > 0.f && lnDl < 0.f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        shadow_prd.inShadow = false;
        const float area = length(cross(light.v1, light.v2));
        Ray shadow_ray =
            make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);

        if (!shadow_prd.inShadow) {
          result += kd_val           // brdf
                    * nDl            // normal attenuation
                    * (-lnDl) * area // visible area
                    * light.emission / Ldist / Ldist * shadow_prd.attenuation;
        }
      }
    }

  } else {
    // transmission
    current_prd.origin = hitpoint;
  }

  current_prd.radiance = result;
}

RT_PROGRAM void any_hit() {
  if (has_kd_map) {
    if (tex2D(kd_map, texcoord.x, texcoord.y).w < 0.1f) {
      rtIgnoreIntersection();
    }
  }
}

RT_PROGRAM void shadow_any_hit() {
  if (!use_shadow) {
    rtTerminateRay();
    return;
  }

  float3 kd_val = make_float3(kd);
  float alpha = kd.w;
  if (has_kd_map) {
    float4 texture_kd_val = tex2D(kd_map, texcoord.x, texcoord.y);
    kd_val = make_float3(texture_kd_val);
    alpha = texture_kd_val.w;
  }

  if (alpha == 1) {
    prd_shadow.inShadow = true;
    rtTerminateRay();
  } else {
    prd_shadow.attenuation *= (1 - alpha);
  }
}
