#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_device.h>
#include "random.h"
#include "prd.h"
#include "light.h"


rtDeclareVariable(rtObject,          top_object,                ,                           );
rtDeclareVariable(rtObject,          top_shadower,              ,                           );
rtDeclareVariable(float,             scene_epsilon,             ,                           );
rtDeclareVariable(unsigned int,      pathtrace_shadow_ray_type, ,                           );

rtBuffer<DirectionalLight>    directional_lights;
rtBuffer<PointLight>          point_lights;
rtBuffer<ParallelogramLight>  parallelogram_lights;

rtDeclareVariable(optix::Ray,        ray,                       rtCurrentRay,               );
rtDeclareVariable(float3,            kd,                        ,                           );
rtDeclareVariable(int,               has_kd_map,                ,                           );
rtDeclareVariable(float3,            texcoord,                  attribute texcoord,         ); 
rtDeclareVariable(float3,            hit_point,                 attribute hit_point,        ); 

rtDeclareVariable(int,               use_shadow,                ,                           );

rtTextureSampler<float4,             2> kd_map;


rtDeclareVariable(PerRayData,        current_prd,               rtPayload,                  );
rtDeclareVariable(PerRayData_shadow, prd_shadow,                rtPayload,                  );
rtDeclareVariable(float3,            geometric_normal,          attribute geometric_normal, ); 
rtDeclareVariable(float3,            shading_normal,            attribute shading_normal,   ); 
rtDeclareVariable(float,             t_hit,                     rtIntersectionDistance,     );


RT_PROGRAM void closest_hit() {
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal               = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
  float3 hitpoint               = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);

  current_prd.max_depth_override = 0;

  float3 kd_val = kd;
  if (has_kd_map) {
    kd_val = make_float3( tex2D( kd_map, texcoord.x, texcoord.y ) );
  }

  current_prd.origin = hitpoint;
  float z1 = rnd(current_prd.seed);
  float z2 = rnd(current_prd.seed);
  float3 p;
  cosine_sample_hemisphere(z1, z2, p);
  Onb onb(ffnormal);
  onb.inverse_transform(p);
  current_prd.direction = p;

  current_prd.attenuation *= kd;

  float3 result = make_float3(0.f);
  for (int i = 0; i < directional_lights.size(); i++) {
    DirectionalLight light = directional_lights[i];
    const float3 L = -light.direction;
    const float nDl = dot(ffnormal, L);
    if (nDl > 0.f) {
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      shadow_prd.inShadow = false;
      Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX);
      rtTrace(top_shadower, shadow_ray, shadow_prd);

      if (!shadow_prd.inShadow) {
        result += nDl * light.emission * kd_val * shadow_prd.attenuation;
      }
    }
  }

  for (int i = 0; i < point_lights.size(); i++) {
    PointLight light = point_lights[i];
    const float  Ldist = length(light.position - hitpoint);
    const float3 L = normalize(light.position - hitpoint);
    const float nDl = dot(ffnormal, L);
    if (nDl > 0.f) {
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      shadow_prd.inShadow = false;
      Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
      rtTrace(top_shadower, shadow_ray, shadow_prd);

      if (!shadow_prd.inShadow) {
        result += kd_val * nDl * light.emission / Ldist / Ldist * shadow_prd.attenuation;
      }
    }
  }

  current_prd.radiance = result;
}

RT_PROGRAM void any_hit() {
  if (has_kd_map) {
    if (tex2D( kd_map, texcoord.x, texcoord.y ).w < 0.1f) {
      rtIgnoreIntersection();
    }
  }
}

RT_PROGRAM void shadow_any_hit() {
  if (!use_shadow) {
    rtTerminateRay();
    return;
  }

  if (has_kd_map) {
    if (tex2D( kd_map, texcoord.x, texcoord.y ).w < 0.1f) {
      rtIgnoreIntersection();
      return;
    }
  }

  prd_shadow.inShadow = true;
  rtTerminateRay();
}
