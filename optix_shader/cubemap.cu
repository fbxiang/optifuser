#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_math.h>

#include "prd.h"

rtDeclareVariable(int, envmapId, , );
rtDeclareVariable(PerRayData, current_prd, rtPayload, );

RT_PROGRAM void miss() {
  current_prd.radiance = make_float3(rtTexCubemap<float4>(
      envmapId, -current_prd.direction.y, current_prd.direction.z, -current_prd.direction.x));
  current_prd.done = 1;
}
