#include "hip/hip_runtime.h"
#include "light.h"
#include "prd.h"
#include "random.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>


rtDeclareVariable(rtObject,          top_object,                ,                           );
rtDeclareVariable(float,             scene_epsilon,             ,                           );
rtDeclareVariable(unsigned int,      pathtrace_shadow_ray_type, ,                           );

rtBuffer<DirectionalLight>    directional_lights;
rtBuffer<PointLight>          point_lights;
rtBuffer<ParallelogramLight>  parallelogram_lights;

rtDeclareVariable(optix::Ray,        ray,                       rtCurrentRay,               );
rtDeclareVariable(float3,            hit_point,                 attribute hit_point,        ); 

rtDeclareVariable(int,               use_shadow,                ,                           );

rtTextureSampler<float4,             2> kd_map;


rtDeclareVariable(PerRayData,        current_prd,               rtPayload,                  );
rtDeclareVariable(PerRayData_shadow, prd_shadow,                rtPayload,                  );
rtDeclareVariable(float3,            geometric_normal,          attribute geometric_normal, ); 
rtDeclareVariable(float3,            shading_normal,            attribute shading_normal,   ); 
rtDeclareVariable(float,             t_hit,                     rtIntersectionDistance,     );


RT_PROGRAM void closest_hit() {
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal               = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
  float3 hitpoint               = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);

  float3 n = normalize(ffnormal);
  float3 d = normalize(-ray.direction);

  current_prd.origin = hitpoint;

  float3 r = reflect(-d, n);
  float3 t = make_float3(0.f, 0.f, 0.f);

  float fr;
  float ni, nt;

  if (dot(world_geometric_normal, d) > 0) {
    // air to water
    ni = 1.0; nt = 1.33;
  } else {
    // water to air
    ni = 1.33; nt = 1.0;
  }
  float3 z = ni / nt * (dot(d, n) * n - d);
  float z2 = dot(z, z);
  if (z2 > 1) {
    // total internal reflection
    fr = 1;
  } else {
    t = normalize(z - sqrt(1 - z2) * n);
    float dnd = dot(n, d);
    float dnt = dot(n, t);
    float rs = (nt * dnd + ni * dnt) / (nt * dnd - ni * dnt);
    float rp = (ni * dnd + nt * dnt) / (ni * dnd - nt * dnt);
    fr = (rs * rs + rp * rp) / 2.f;
  }

  float3 p;
  if (rnd(current_prd.seed) < fr) {
    // reflect
    p = r;
  } else {
    // refract
    p = t;
    current_prd.attenuation *= make_float3(0.8, 0.9, 1);
  }

  current_prd.max_depth_override = 7;
  current_prd.direction = p;
  current_prd.radiance = make_float3(0.f);
}

RT_PROGRAM void any_hit() {
}

RT_PROGRAM void shadow_any_hit() {
  if (!use_shadow) {
    rtTerminateRay();
    return;
  }

  prd_shadow.attenuation = make_float3(0.8, 0.9, 1.0);
}
