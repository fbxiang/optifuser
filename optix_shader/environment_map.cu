#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_math.h>

#include "prd.h"

rtTextureSampler<float4, 2> envmap;
rtDeclareVariable(PerRayData, current_prd, rtPayload, );

RT_PROGRAM void miss() {
  float theta = atan2f(current_prd.direction.x, current_prd.direction.z);
  float phi = M_PIf * 0.5f - acosf(current_prd.direction.y);
  float u = (theta + M_PIf) * (0.5f * M_1_PIf);
  float v = 0.5f * (1.0f + sin(phi));
  current_prd.result = make_float3(tex2D(envmap, u, v));
  current_prd.done = 1;
}
