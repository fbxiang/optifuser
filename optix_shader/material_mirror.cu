#include "hip/hip_runtime.h"
#include "light.h"
#include "prd.h"
#include "random.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>


rtDeclareVariable(rtObject,          top_object,                ,                           );
rtDeclareVariable(float,             scene_epsilon,             ,                           );
rtDeclareVariable(unsigned int,      pathtrace_shadow_ray_type, ,                           );

rtBuffer<DirectionalLight>    directional_lights;
rtBuffer<PointLight>          point_lights;
rtBuffer<ParallelogramLight>  parallelogram_lights;

rtDeclareVariable(optix::Ray,        ray,                       rtCurrentRay,               );
rtDeclareVariable(float3,            hit_point,                 attribute hit_point,        ); 

rtDeclareVariable(int,               use_shadow,                ,                           );

rtTextureSampler<float4,             2> kd_map;


rtDeclareVariable(PerRayData,        current_prd,               rtPayload,                  );
rtDeclareVariable(PerRayData_shadow, prd_shadow,                rtPayload,                  );
rtDeclareVariable(float3,            geometric_normal,          attribute geometric_normal, ); 
rtDeclareVariable(float3,            shading_normal,            attribute shading_normal,   ); 
rtDeclareVariable(float,             t_hit,                     rtIntersectionDistance,     );


RT_PROGRAM void closest_hit() {
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal               = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
  float3 hitpoint               = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);

  current_prd.origin = hitpoint;
  float3 p = reflect(normalize(ray.direction), normalize(ffnormal));
  current_prd.direction = p;
  current_prd.attenuation *= 1;
  current_prd.max_depth_override = 2;
  current_prd.radiance = make_float3(0.f);
}

RT_PROGRAM void any_hit() {
}

RT_PROGRAM void shadow_any_hit() {
  if (!use_shadow) {
    rtTerminateRay();
    return;
  }

  prd_shadow.inShadow = true;
  rtTerminateRay();
}
