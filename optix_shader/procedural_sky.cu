#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_math.h>

#include "prd.h"

rtDeclareVariable(PerRayData, current_prd, rtPayload, );

__device__ float3 procedural_color(float angle) {
  float factor = 1.f;
  float3 horizonColor = factor * make_float3(0.9, 0.9, 0.9);
  float3 zenithColor = factor * make_float3(0.522, 0.757, 0.914);
  float3 groundColor = factor * make_float3(0.5, 0.410, 0.271);

  return lerp(lerp(zenithColor, horizonColor, smoothstep(15.f, 5.f, angle)), groundColor,
             smoothstep(-5.f, -15.f, angle));
}

RT_PROGRAM void miss() {
  float phi = asinf(current_prd.direction.z);
  current_prd.radiance = procedural_color(phi * 57.3);
  current_prd.done = 1;
}
