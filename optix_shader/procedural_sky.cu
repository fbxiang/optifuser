#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <optix_math.h>

#include "prd.h"

rtDeclareVariable(PerRayData, current_prd, rtPayload, );

__device__ float3 procedural_color(float angle) {
  float3 horizonColor = .4f * make_float3(0.9, 0.9, 0.9);
  float3 zenithColor = .4f * make_float3(0.522, 0.757, 0.914);
  float3 groundColor = .4f * make_float3(0.5, 0.410, 0.271);

  return lerp(lerp(zenithColor, horizonColor, smoothstep(15.f, 5.f, angle)), groundColor,
             smoothstep(-5.f, -15.f, angle));
}

RT_PROGRAM void miss() {
  float phi = asinf(current_prd.direction.z);
  current_prd.result = procedural_color(phi * 57.3);
  current_prd.done = 1;
}
