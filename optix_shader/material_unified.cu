#include "hip/hip_runtime.h"
#include "light.h"
#include "prd.h"
#include "random.h"
#include "shading_models.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, , );

rtBuffer<DirectionalLight> directional_lights;
rtBuffer<PointLight> point_lights;
rtBuffer<ParallelogramLight> parallelogram_lights;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float4, kd, , );
rtDeclareVariable(float4, ks, , );
rtDeclareVariable(int, has_kd_map, , );
rtDeclareVariable(int, has_ks_map, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, hit_point, attribute hit_point, );

rtDeclareVariable(int, use_shadow, , );

rtTextureSampler<float4, 2> kd_map;
rtTextureSampler<float4, 2> ks_map;

rtDeclareVariable(PerRayData, current_prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void closest_hit() {
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal =
      normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);
  float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);

  current_prd.max_depth_override = 0;

  float3 kd_val = make_float3(kd);
  float transmission = kd.w;
  if (has_kd_map) {
    float4 texture_kd_val = tex2D(kd_map, texcoord.x, texcoord.y);
    kd_val = make_float3(texture_kd_val);
    transmission = texture_kd_val.w;
  }

  float3 ks_val = make_float3(ks);
  float roughness = ks.w;
  if (has_ks_map) {
    float4 texture_ks_val = tex2D(ks_map, texcoord.x, texcoord.y);
    ks_val = make_float3(texture_ks_val);
    roughness = texture_ks_val.w;
  }

  float3 result = make_float3(0.f);
  if (rnd(current_prd.seed) < transmission) {
    current_prd.origin = hitpoint;

    float kd_l = luminance(kd_val);
    float ks_l = luminance(ks_val);

    float3 wi, reflectance;
    if (rnd(current_prd.seed) < kd_l / (kd_l + ks_l)) {
      // lambertian
      SampleDiffuse(-ray.direction, ffnormal, kd_val, wi, reflectance, current_prd.seed);
    } else {
      // GGX
      SampleGGX_ImpD(-ray.direction, ffnormal, roughness, ks_val, wi, reflectance, current_prd.seed);
    }
    current_prd.direction = wi;
    current_prd.attenuation *= reflectance;

    for (int i = 0; i < directional_lights.size(); i++) {
      DirectionalLight light = directional_lights[i];
      const float3 L = -light.direction;
      float3 reflectance = ForwardDiffuse(L, -ray.direction, ffnormal, kd_val) +
                           ForwardGGX(L, -ray.direction, ffnormal, roughness, ks_val);
      if (reflectance.x > 0.f && reflectance.y > 0.f && reflectance.z > 0.f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        shadow_prd.inShadow = false;
        Ray shadow_ray =
            make_Ray(hitpoint, -light.direction, pathtrace_shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_shadower, shadow_ray, shadow_prd);

        if (!shadow_prd.inShadow) {
          result += light.emission * reflectance * shadow_prd.attenuation;
        }
      }
    }

    for (int i = 0; i < point_lights.size(); i++) {
      PointLight light = point_lights[i];
      const float Ldist = length(light.position - hitpoint);
      const float3 L = normalize(light.position - hitpoint);
      float3 reflectance = ForwardDiffuse(L, -ray.direction, ffnormal, kd_val) +
                           ForwardGGX(L, -ray.direction, ffnormal, roughness, ks_val);
      if (reflectance.x > 0.f && reflectance.y > 0.f && reflectance.z > 0.f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        shadow_prd.inShadow = false;
        Ray shadow_ray =
            make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);

        if (!shadow_prd.inShadow) {
          result += reflectance * light.emission / Ldist / Ldist * shadow_prd.attenuation;
        }
      }
    }

    for (int i = 0; i < parallelogram_lights.size(); ++i) {
      ParallelogramLight light = parallelogram_lights[i];

      // sample a point on the light
      float r1 = rnd(current_prd.seed);
      float r2 = rnd(current_prd.seed);
      float3 light_position = light.corner + r1 * light.v1 + r2 * light.v2;
      const float Ldist = length(light_position - hitpoint);
      const float3 L = normalize(light_position - hitpoint);
      const float lnDl = dot(light.normal, L); // light normal dot light direction

      if (lnDl < 0.f) {
        float3 reflectance = ForwardDiffuse(L, -ray.direction, ffnormal, kd_val) +
                             ForwardGGX(L, -ray.direction, ffnormal, roughness, ks_val);
        if (reflectance.x > 0.f && reflectance.y > 0.f && reflectance.z > 0.f) {
          PerRayData_shadow shadow_prd;
          shadow_prd.attenuation = make_float3(1.0f);
          shadow_prd.inShadow = false;
          const float area = length(cross(light.v1, light.v2));
          Ray shadow_ray =
              make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
          rtTrace(top_shadower, shadow_ray, shadow_prd);

          if (!shadow_prd.inShadow) {
            result +=  reflectance
                      * (-lnDl) * area // visible area
                      * light.emission / Ldist / Ldist * shadow_prd.attenuation;
          }
        }
      }
    }

  } else {
    // transmission
    current_prd.origin = hitpoint;
  }

  current_prd.radiance = result;
}

RT_PROGRAM void any_hit() {
  if (has_kd_map) {
    if (tex2D(kd_map, texcoord.x, texcoord.y).w < 0.1f) {
      rtIgnoreIntersection();
    }
  }
}

RT_PROGRAM void shadow_any_hit() {
  if (!use_shadow) {
    rtTerminateRay();
    return;
  }

  float3 kd_val = make_float3(kd);
  float alpha = kd.w;
  if (has_kd_map) {
    float4 texture_kd_val = tex2D(kd_map, texcoord.x, texcoord.y);
    kd_val = make_float3(texture_kd_val);
    alpha = texture_kd_val.w;
  }

  if (alpha == 1) {
    prd_shadow.inShadow = true;
    rtTerminateRay();
  } else {
    prd_shadow.attenuation *= (1 - alpha);
  }
}
