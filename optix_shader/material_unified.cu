#include "hip/hip_runtime.h"
#include "light.h"
#include "prd.h"
#include "random.h"
#include "shading_models.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(unsigned int, pathtrace_shadow_ray_type, , );

rtBuffer<DirectionalLight> directional_lights;
rtBuffer<PointLight> point_lights;
rtBuffer<ParallelogramLight> parallelogram_lights;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float4, kd, , );
rtDeclareVariable(float, ks, , );
rtDeclareVariable(int, has_kd_map, , );
rtDeclareVariable(int, has_ks_map, , );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, hit_point, attribute hit_point, );

rtDeclareVariable(float, roughness, , );
rtDeclareVariable(float, metallic, , );

rtDeclareVariable(int, use_shadow, , );

rtTextureSampler<float4, 2> kd_map;
rtTextureSampler<float4, 2> ks_map;

rtDeclareVariable(PerRayData, current_prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void closest_hit() {
  float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal =
      normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 wo = -ray.direction;
  float3 ffnormal = faceforward(world_shading_normal, wo, world_geometric_normal);
  float3 hitpoint = rtTransformPoint(RT_OBJECT_TO_WORLD, hit_point);

  current_prd.max_depth_override = 0;

  float3 kd_val = make_float3(kd);
  float transmission = kd.w;
  if (has_kd_map) {
    float4 texture_kd_val = tex2D(kd_map, texcoord.x, texcoord.y);
    kd_val = make_float3(texture_kd_val);
    transmission = texture_kd_val.w;
  }

  float ks_val = ks;
  if (has_ks_map) {
    float4 texture_ks_val = tex2D(ks_map, texcoord.x, texcoord.y);
    ks_val = texture_ks_val.x;
  }

  float3 radiance = make_float3(0.f);
  // if (rnd(current_prd.seed) < transmission) {
  if (rnd(current_prd.seed) < transmission) {
    current_prd.origin = hitpoint;

    float kd_l = fmaxf(0.f, luminance(kd_val));
    float ks_l = fmaxf(0.f, fminf(ks_val, 1.f));

    float3 wi, reflectance;

    float pd = kd_l / fmaxf(kd_l + ks_l, 0.001);
    if (rnd(current_prd.seed) < pd) {
      // sample diffuse-metallic
      if (rnd(current_prd.seed) < metallic) {
        // metal
        SampleGGX_ImpD(wo, ffnormal, roughness, 1.f, kd_val, wi, reflectance, current_prd.seed);
      } else {
        // diffuse
        SampleDiffuse(wo, ffnormal, kd_val, wi, reflectance, current_prd.seed);
      }
      reflectance /= pd; // importance sampling
    } else {
      // sample specular
      SampleGGX_ImpD(wo, ffnormal, roughness, ks_val, make_float3(1.f), wi, reflectance,
                     current_prd.seed);
      reflectance /= (1 - pd); // importance sampling
    }

    if (current_prd.depth == 0) {
      current_prd.albedo = kd_val;
      current_prd.normal = ffnormal;
    }
    current_prd.direction = wi;
    current_prd.attenuation *= reflectance;

    for (int i = 0; i < directional_lights.size(); i++) {
      DirectionalLight light = directional_lights[i];
      const float3 L = -normalize(light.direction);
      float3 reflectance = (1 - metallic) * ForwardDiffuse(L, wo, ffnormal, kd_val);
      reflectance += metallic * ForwardGGX(L, wo, ffnormal, roughness, 1.f, kd_val);
      reflectance += ForwardGGX(L, wo, ffnormal, roughness, ks_val, make_float3(1.f));

      if (reflectance.x > 0.f || reflectance.y > 0.f || reflectance.z > 0.f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        shadow_prd.inShadow = false;
        Ray shadow_ray =
            make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX);
        rtTrace(top_shadower, shadow_ray, shadow_prd);

        if (!shadow_prd.inShadow) {
          radiance += light.emission * reflectance * shadow_prd.attenuation;
        }
      }
    }

    for (int i = 0; i < point_lights.size(); i++) {
      PointLight light = point_lights[i];
      const float Ldist = length(light.position - hitpoint);
      const float3 L = normalize(light.position - hitpoint);
      float3 reflectance = (1 - metallic) * ForwardDiffuse(L, wo, ffnormal, kd_val);
      reflectance += metallic * ForwardGGX(L, wo, ffnormal, roughness, 1.f, kd_val);
      reflectance += ForwardGGX(L, wo, ffnormal, roughness, ks_val, make_float3(1.f));

      if (reflectance.x > 0.f || reflectance.y > 0.f || reflectance.z > 0.f) {
        PerRayData_shadow shadow_prd;
        shadow_prd.attenuation = make_float3(1.0f);
        shadow_prd.inShadow = false;
        Ray shadow_ray =
            make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon);
        rtTrace(top_shadower, shadow_ray, shadow_prd);

        if (!shadow_prd.inShadow) {
          radiance += reflectance * light.emission / Ldist / Ldist * shadow_prd.attenuation;
        }
      }
    }

    for (int i = 0; i < parallelogram_lights.size(); ++i) {
      ParallelogramLight light = parallelogram_lights[i];

      // sample a point on the light
      float r1 = rnd(current_prd.seed);
      float r2 = rnd(current_prd.seed);
      float3 light_position = light.corner + r1 * light.v1 + r2 * light.v2;
      const float Ldist = length(light_position - hitpoint);
      const float3 L = normalize(light_position - hitpoint);
      const float lnDl = dot(light.normal, L); // light normal dot light direction

      if (lnDl < 0.f) {
        float3 reflectance = (1 - metallic) * ForwardDiffuse(L, wo, ffnormal, kd_val);
        reflectance += metallic * ForwardGGX(L, wo, ffnormal, roughness, 1.f, kd_val);
        reflectance += ForwardGGX(L, wo, ffnormal, roughness, ks_val, make_float3(1.f));
        if (reflectance.x > 0.f || reflectance.y > 0.f || reflectance.z > 0.f) {
          PerRayData_shadow shadow_prd;
          shadow_prd.attenuation = make_float3(1.0f);
          shadow_prd.inShadow = false;
          const float area = length(cross(light.v1, light.v2));
          Ray shadow_ray = make_Ray(hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon,
                                    Ldist - scene_epsilon);
          rtTrace(top_shadower, shadow_ray, shadow_prd);

          if (!shadow_prd.inShadow) {
            radiance += reflectance * (-lnDl) * area // visible area
                        * light.emission / Ldist / Ldist * shadow_prd.attenuation;
          }
        }
      }
    }

  } else {
    // transmission
    current_prd.origin = hitpoint;
  }

  current_prd.radiance = radiance;
}

RT_PROGRAM void any_hit() {
  if (has_kd_map) {
    if (tex2D(kd_map, texcoord.x, texcoord.y).w < 0.1f) {
      rtIgnoreIntersection();
    }
  }
}

RT_PROGRAM void shadow_any_hit() {
  if (!use_shadow) {
    rtTerminateRay();
    return;
  }

  float3 kd_val = make_float3(kd);
  float alpha = kd.w;
  if (has_kd_map) {
    float4 texture_kd_val = tex2D(kd_map, texcoord.x, texcoord.y);
    kd_val = make_float3(texture_kd_val);
    alpha = texture_kd_val.w;
  }

  if (alpha == 1) {
    prd_shadow.inShadow = true;
    rtTerminateRay();
  } else {
    prd_shadow.attenuation *= (1 - alpha);
  }
}
